#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>//needed to save output as a bitmap file.
#include <stdio.h>

const char inputpath[] = { "D:\\HGT2\\S40E175.hgt" };//source patrh for the HGT file
const char outputfile[] = { "D:\\HGT2\\_Output\\NormalMapCUDA.bmp" };//path to dump the output file (normal map)

const short HGT_DIM = 3601;//resolution of HGT files (1 arc-second)
const int NORM_DIM = 3600;//resolution of normal map. 
//Note HGT files are  conveniently 3601 so we don't have problems loading adjadaent HGT files to get the correct values at the border.
const float NORM_DIM_F = 3600.0f;
const int NormalMapSize = NORM_DIM * NORM_DIM;
const int arraySize = HGT_DIM * HGT_DIM;
 
//--------------------------------------------------------------------------------//
//function to save as bitmap (Windows only)
bool SaveBitmapRGB(BYTE* Buffer, int width, int height, long paddedsize, LPCTSTR bmpfile)
{
    BITMAPFILEHEADER bmfh;
    BITMAPINFOHEADER info;
    memset(&bmfh, 0, sizeof(BITMAPFILEHEADER));
    memset(&info, 0, sizeof(BITMAPINFOHEADER));

    bmfh.bfType = 0x4d42;       // 0x4d42 = 'BM'
    bmfh.bfReserved1 = 0;
    bmfh.bfReserved2 = 0;
    bmfh.bfSize = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + paddedsize;
    bmfh.bfOffBits = 0x36;

    info.biSize = sizeof(BITMAPINFOHEADER);
    info.biWidth = width;
    info.biHeight = height;
    info.biPlanes = 1;
    info.biBitCount = 24;
    info.biCompression = BI_RGB;
    info.biSizeImage = 0;
    info.biXPelsPerMeter = 0x0ec4;
    info.biYPelsPerMeter = 0x0ec4;
    info.biClrUsed = 0;
    info.biClrImportant = 0;
    HANDLE file = CreateFile(bmpfile, GENERIC_WRITE, FILE_SHARE_READ, NULL, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL, NULL);
    if (NULL == file)
    {
        CloseHandle(file);
        return false;
    }

    unsigned long bwritten;
    if (WriteFile(file, &bmfh, sizeof(BITMAPFILEHEADER),
        &bwritten, NULL) == false)
    {
        CloseHandle(file);
        return false;
    }

    if (WriteFile(file, &info, sizeof(BITMAPINFOHEADER),
        &bwritten, NULL) == false)
    {
        CloseHandle(file);
        return false;
    }

    if (WriteFile(file, Buffer, paddedsize, &bwritten, NULL) == false)
    {
        CloseHandle(file);
        return false;
    }

    CloseHandle(file);
    return true;
}
//--------------------------------------------------------------------------------//
hipError_t HGTtoNormalCuda(float3*c, const short*a, unsigned int size, unsigned int normalmapsize);
//--------------------------------------------------------------------------------//
// Kernel Helper functions
__device__  float3 normalize(float3 v)
{
    double len = sqrt((float)(v.x * v.x + v.y * v.y + v.z * v.z));
    v.x /= len;
    v.y /= len;
    v.z /= len;
    return v;
}
//--------------------------------------------------------------------------------//
__device__ float3 GetNormal(float p1x, float p1y, float p1z, float p2x, float p2y, float p2z, float p3x, float p3y, float p3z)
{
    long nScale = 30;//approximately 30 meters per point for high resolution HGT files (90 when using the low res HGT format)
    p1x = p1x * nScale;
    p1y = p1y * nScale;
    p2x = p2x * nScale;
    p2y = p2y * nScale;
    p3x = p3x * nScale;
    p3y = p3y * nScale;
    float Ax = p2x - p1x;
    float Ay = p2y - p1y;
    float Az = p2z - p1z;
    float Bx = p3x - p1x;
    float By = p3y - p1y;
    float Bz = p3z - p1z;
    float3 n;
    n.x = Ay * Bz - Az * By;
    n.y = Az * Bx - Ax * Bz;
    n.z = Ax * By - Ay * Bx;
    n = normalize(n);
    return n;
}
//--------------------------------------------------------------------------------//
__device__ float GetHeight(const short* a, int h, int j)
{
    int tid = j * HGT_DIM + h;
    return (float)a[tid];
}
//--------------------------------------------------------------------------------//
//main Kernal
__global__ void HGTToNormalKernel(float3*c, const short*a, int count)
{  
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
    int threadPosInBlock = threadIdx.x + 
        blockDim.x * threadIdx.y +
        blockDim.x * blockDim.y * threadIdx.z;
    int blockPosInGrid = blockIdx.x +
        gridDim.x * blockIdx.y +
        gridDim.x * gridDim.y * blockIdx.z;
    int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;//calulcate global indiex to array
    if (tid < count)
    {  
        int h = tid % NORM_DIM;
        int j = tid / NORM_DIM;
        //calulcate the normal for the two adjacent triangles in this cell and average them
        float3 v3a = GetNormal(h, j, GetHeight(a, h, j), h + 1, j, GetHeight(a, h + 1, j), h, j + 1, GetHeight(a, h, j + 1));
        float3 v3b = GetNormal(h + 1, j, GetHeight(a, h + 1, j), h + 1, j + 1, GetHeight(a, h + 1, j + 1), h, j + 1, GetHeight(a, h, j + 1) );
        float3 vNornmal;
        vNornmal.x = (v3a.x + v3b.x) / 2;
        vNornmal.y = (v3a.y + v3b.y) / 2;
        vNornmal.z = (v3a.z + v3b.z) / 2;
        c[tid] = normalize(vNornmal); 
    }
}
//-----------------------------------------------------------------------------------// 
int main()
{    
    short* pHGTData = new short[arraySize]; 
    float3* pNormData = new float3[NormalMapSize];

    //load HGT file and reverse the byte order
    FILE* pFile = 0;
    pFile = fopen(inputpath, "rb");
    if (pFile != 0)
    {
        short i = 0;
        while (true)
        {
            int n = fread((char*)(pHGTData + i * HGT_DIM), sizeof(short), HGT_DIM, pFile);
            i++;
            if (n == 0) break;
        }
        fclose(pFile);

        for (int h = 0; h < HGT_DIM * HGT_DIM; h++)
        {
            short w = pHGTData[h];
            pHGTData[h] = MAKEWORD(HIBYTE(w), LOBYTE(w));
        }


        //Calulcate the normal map.
        hipError_t cudaStatus = HGTtoNormalCuda(pNormData, pHGTData, arraySize, NormalMapSize);
        if (cudaStatus == hipSuccess) {

            printf(" c[0].xyz = {%f,%f,%f}\n", pNormData[0].x, pNormData[1].y, pNormData[2].z);
            //save as a bitmap to view the normals. Normals are in Tangent space
            BYTE* pBMPData = new BYTE[NORM_DIM * NORM_DIM * 3];
            for (int h = 0; h < NORM_DIM; h++)
            {
                for (int j = 0; j < NORM_DIM; j++)
                {
                    float3 normal = pNormData[h * NORM_DIM + j];
                    pBMPData[(NORM_DIM - h - 1) * NORM_DIM * 3 + j * 3 + 0] = 255 * (0.5 + 0.5 * normal.z);
                    pBMPData[(NORM_DIM - h - 1) * NORM_DIM * 3 + j * 3 + 1] = 255 * (0.5 + 0.5 * -1 * normal.x);//invert green axis
                    pBMPData[(NORM_DIM - h - 1) * NORM_DIM * 3 + j * 3 + 2] = 255 * (0.5 + 0.5 * normal.y);
                    //hmmm is red and green reversed?? Had to swap y and x around...
                }
            }
            SaveBitmapRGB(pBMPData, NORM_DIM, NORM_DIM, NORM_DIM * NORM_DIM * 3, outputfile);
            delete[] pBMPData;

            // hipDeviceReset must be called before exiting in order for profiling and
            // tracing tools such as Nsight and Visual Profiler to show complete traces.
            cudaStatus = hipDeviceReset();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceReset failed!");
                return 1;
            }
        }
        else
        {
            fprintf(stderr, "HGTtoNormalCuda failed!");
        }
    }
    delete[] pHGTData;
    delete[] pNormData;

    return 0;
}
//-----------------------------------------------------------------------------------// 
// Helper function for using CUDA to caluclate normal map fro high res HGT file in parallel. 
hipError_t HGTtoNormalCuda(float3 * pNormData, const short* pHGTData,  unsigned int size, unsigned int NormalMapSize)
{
    short *devHGTData = 0;
    float3 *devNormData = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&devNormData, NormalMapSize * sizeof(float3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&devHGTData, size * sizeof(short));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(devHGTData, pHGTData, size * sizeof(short), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    // Launch a kernel on the GPU with one thread for each element.
    const int count = NORM_DIM * NORM_DIM;
    dim3 block(8, 8, 8);
    dim3 grid(450, 450);
    HGTToNormalKernel<<<grid, block>>>(devNormData, devHGTData, count);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "HGTToNormalKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pNormData, devNormData, NormalMapSize * sizeof(float3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
  
Error:
    hipFree(devNormData);
    hipFree(devHGTData);
    return cudaStatus;
}
//-----------------------------------------------------------------------------------// 